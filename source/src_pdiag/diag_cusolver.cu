#include "hip/hip_runtime.h"
#include "diag_cusolver.cuh"
#include "nvToolsExt.h"
#include <hip/hip_runtime.h>
#include <hipsolver.h>

void printMatrix(int m, int n, const double *A, int lda, const char* name)
{
    for(int row = 0 ; row < m ; row++){        // row dominant  
        for(int col = 0 ; col < n ; col++){
            double Areg = A[row + col*lda];
            printf("%s(%d,%d) = ", name, row+1, col+1);
            std::cout << Areg << std::endl;
             
        }
    } 
}

int cusolver_DnDsygvd(int N, int M, double *A, double *B, double *W, double *V)
{
    printf("enter begin: N=  %d M = %d \n", N, M);
    hipEvent_t start_all, stop_all;
    hipEventCreate(&start_all);
    hipEventCreate(&stop_all);
    hipEventRecord(start_all, 0);

    hipsolverHandle_t cusolverH = NULL;
    hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess;
    hipError_t cudaStat2 = hipSuccess;
    hipError_t cudaStat3 = hipSuccess;
    hipError_t cudaStat4 = hipSuccess;

    
    int m = N;
    int lda = m;

    double *d_A = NULL;
    double *d_B = NULL;

    double *d_W = NULL;
    int *devInfo = NULL;
    double *d_work = NULL;
    int  lwork = 0;
    int info_gpu = 0;
    // printf("A = (matlab base-1)\n");
    // printMatrix(m, m, A, lda, "A");
    // printf("=====\n");
    // printf("B = (matlab base-1)\n");
    // printMatrix(m, m, B, lda, "B");
    // printf("=====\n");

    // step 1: create cusolver/cublas handle
    cusolver_status = hipsolverDnCreate(&cusolverH);
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
    // step 2: copy A and B to device
    cudaStat1 = hipMalloc ((void**)&d_A, sizeof(double) * lda * m);
    cudaStat2 = hipMalloc ((void**)&d_B, sizeof(double) * lda * m);
    cudaStat3 = hipMalloc ((void**)&d_W, sizeof(double) * m);
    cudaStat4 = hipMalloc ((void**)&devInfo, sizeof(int));
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);
    assert(hipSuccess == cudaStat4);
    cudaStat1 = hipMemcpy(d_A, A, sizeof(double) * lda * m, hipMemcpyHostToDevice);
    cudaStat2 = hipMemcpy(d_B, B, sizeof(double) * lda * m, hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);

    // step 3: query working space of sygvd
    hipsolverEigType_t itype = HIPSOLVER_EIG_TYPE_1; // A*x = (lambda)*B*x
    hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; // compute eigenvalues and eigenvectors.
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
    //The helper functions below can calculate the sizes needed for pre-allocated buffer.
    //The S and D data types are real valued single and double precision, respectively.
    // The C and Z data types are complex valued single and double precision, respectively.
    cusolver_status = hipsolverDnDsygvd_bufferSize(        
        cusolverH,
        itype,
        jobz,
        uplo,
        m,
        d_A,
        lda,
        d_B,
        lda,
        d_W,
        &lwork);
    assert (cusolver_status == HIPSOLVER_STATUS_SUCCESS);
    cudaStat1 = hipMalloc((void**)&d_work, sizeof(double)*lwork);
    assert(hipSuccess == cudaStat1);

    // step 4: compute spectrum of (A,B)
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    cusolver_status = hipsolverDnDsygvd(
        cusolverH,
        itype,
        jobz,
        uplo,
        m,
        d_A,
        lda,
        d_B,
        lda,
        d_W,
        d_work,
        lwork,
        devInfo);
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);     
    printf("time=%f ms\n",elapsedTime);

    cudaStat1 = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
    assert(hipSuccess == cudaStat1);

    cudaStat1 = hipMemcpy(W, d_W, sizeof(double)*m, hipMemcpyDeviceToHost);
    cudaStat2 = hipMemcpy(V, d_A, sizeof(double)*lda*m, hipMemcpyDeviceToHost);
    cudaStat3 = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);
    // printf("after sygvd: info_gpu = %d\n", info_gpu);
    assert(0 == info_gpu);
    // printf("eigenvalue = (matlab base-1), ascending order\n");
    // for(int i = 0 ; i < min(N,10) ; i++){
    //     printf("W[%d] = %E\n", i+1, W[i]);
    // }
    // for(int i = max(0, K-10) ; i < K ; i++){
    //     printf("W[%d] = %E\n", i+1, W[i]);
    // }
    // printf("V = (matlab base-1)\n");
    // printMatrix(m, m, V, lda, "V");
    // printf("=====\n");
    // step 4: check eigenvalues
    // double lambda_sup = 0;
    // for(int i = 0 ; i < m ; i++){
    //     double error = fabs( lambda[i] - W[i]);
    // }   lambda_sup = (lambda_sup > error)? lambda_sup : error;
    // printf("|lambda - W| = %E\n", lambda_sup);

    // free resources
    if (d_A    ) hipFree(d_A);
    if (d_B    ) hipFree(d_B);
    if (d_W    ) hipFree(d_W);
    if (devInfo) hipFree(devInfo);
    if (d_work ) hipFree(d_work);
    if (cusolverH) hipsolverDnDestroy(cusolverH);

    hipEventRecord(stop_all, 0);
    hipEventSynchronize(stop_all);
    float elapsedTime_all;
    hipEventElapsedTime(&elapsedTime_all, start_all, stop_all);
    hipEventDestroy(start_all);
    hipEventDestroy(stop_all);     
    printf("all_time=%f ms\n",elapsedTime_all);

    hipDeviceReset();
    return 0; 
}



int cusolver_DnZhegvd(int N, int M, std::complex<double>  *A, std::complex<double>  *B, double *W, std::complex<double>  *V)
{
    printf("enter begin: N=  %d M = %d \n", N, M);
    hipsolverHandle_t cusolverH = NULL;
    hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess;
    hipError_t cudaStat2 = hipSuccess;
    hipError_t cudaStat3 = hipSuccess;
    hipError_t cudaStat4 = hipSuccess;

    
    int m = N;
    int lda = m;

    hipDoubleComplex *d_A = NULL;
    hipDoubleComplex *d_B = NULL;

    double *d_W = NULL;
    int *devInfo = NULL;
    hipDoubleComplex *d_work = NULL;
    int  lwork = 0;
    int info_gpu = 0;
    // printf("A = (matlab base-1)\n");
    // printMatrix(m, m, A, lda, "A");
    // printf("=====\n");
    // printf("B = (matlab base-1)\n");
    // printMatrix(m, m, B, lda, "B");
    // printf("=====\n");

    // step 1: create cusolver/cublas handle
    cusolver_status = hipsolverDnCreate(&cusolverH);
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
    // step 2: copy A and B to device
    cudaStat1 = hipMalloc ((void**)&d_A, sizeof(hipDoubleComplex) * lda * m);
    cudaStat2 = hipMalloc ((void**)&d_B, sizeof(hipDoubleComplex) * lda * m);
    cudaStat3 = hipMalloc ((void**)&d_W, sizeof(double) * m);
    cudaStat4 = hipMalloc ((void**)&devInfo, sizeof(int));
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);
    assert(hipSuccess == cudaStat4);
    cudaStat1 = hipMemcpy(d_A, A, sizeof(hipDoubleComplex) * lda * m, hipMemcpyHostToDevice);
    cudaStat2 = hipMemcpy(d_B, B, sizeof(hipDoubleComplex) * lda * m, hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);

    // step 3: query working space of sygvd
    hipsolverEigType_t itype = HIPSOLVER_EIG_TYPE_1; // A*x = (lambda)*B*x
    hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; // compute eigenvalues and eigenvectors.
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
    //The helper functions below can calculate the sizes needed for pre-allocated buffer.
    //The S and D data types are real valued single and double precision, respectively.
    // The C and Z data types are complex valued single and double precision, respectively.
    cusolver_status = hipsolverDnZhegvd_bufferSize(        
        cusolverH,
        itype,
        jobz,
        uplo,
        m,
        d_A,
        lda,
        d_B,
        lda,
        d_W,
        &lwork);
    assert (cusolver_status == HIPSOLVER_STATUS_SUCCESS);
    cudaStat1 = hipMalloc((void**)&d_work, sizeof(hipDoubleComplex)*lwork);
    assert(hipSuccess == cudaStat1);

    // step 4: compute spectrum of (A,B)
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    cusolver_status = hipsolverDnZhegvd(
        cusolverH,
        itype,
        jobz,
        uplo,
        m,
        d_A,
        lda,
        d_B,
        lda,
        d_W,
        d_work,
        lwork,
        devInfo);
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("time=%f\n",elapsedTime);

    cudaStat1 = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
    assert(hipSuccess == cudaStat1);

    cudaStat1 = hipMemcpy(W, d_W, sizeof(double)*m, hipMemcpyDeviceToHost);
    cudaStat2 = hipMemcpy(V, d_A, sizeof(hipDoubleComplex)*lda*m, hipMemcpyDeviceToHost);
    cudaStat3 = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);
    // printf("after sygvd: info_gpu = %d\n", info_gpu);
    assert(0 == info_gpu);
    // printf("eigenvalue = (matlab base-1), ascending order\n");
    // for(int i = 0 ; i < min(N,10) ; i++){
    //     printf("W[%d] = %E\n", i+1, W[i]);
    // }
    // for(int i = max(0, N-10) ; i < N ; i++){
    //     printf("W[%d] = %E\n", i+1, W[i]);
    // }
    // printf("V = (matlab base-1)\n");
    // printMatrix(m, m, V, lda, "V");
    // printf("=====\n");
    // step 4: check eigenvalues
    // double lambda_sup = 0;
    // for(int i = 0 ; i < m ; i++){
    //     double error = fabs( lambda[i] - W[i]);
    // }   lambda_sup = (lambda_sup > error)? lambda_sup : error;
    // printf("|lambda - W| = %E\n", lambda_sup);

    // free resources
    if (d_A    ) hipFree(d_A);
    if (d_B    ) hipFree(d_B);
    if (d_W    ) hipFree(d_W);
    if (devInfo) hipFree(devInfo);
    if (d_work ) hipFree(d_work);
    if (cusolverH) hipsolverDnDestroy(cusolverH);
    hipDeviceReset();
    return 0; 
}

